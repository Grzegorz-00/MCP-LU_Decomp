#include "hip/hip_runtime.h"
#include "Solver.h"


__global__ void computeRow(double* aPtr, int i, int size, int* error)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if(*error != 0)return;
	int j = idx+i;
	if(j >= i && j<size)
	{
		double sum = 0;
		for(int k = 0;k<i;k++)
		{
			sum+=aPtr[k*size + i]*aPtr[j*size + k];
		}
		aPtr[j*size + i] -= sum;
	}
}
__global__ void computeCol(double* aPtr, int i, int size, int* error)
{
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if(*error != 0)return;
	int j = idx+i;

	if(j >= i+1 && j<size)
	{
		double sum = 0;
		for(int k = 0;k<i;k++)
		{
			sum+=aPtr[k*size + j]*aPtr[i*size + k];
		}
		aPtr[i*size + j] -=sum;
		if(aPtr[i*size + i] == 0)
		{
			*error = 1;
			return;
		}
		aPtr[i*size + j] = aPtr[i*size + j]/aPtr[i*size + i];

	}
	//return;
}

int Solver::LUSolveCPU(Matrix& A, Vector& x, Vector& b)
{
	int size = A.getSize();
	double** temp = A.getPtr();
	double** aPtr = new double*[size];
	for(int i = 0;i<size;i++)
	{
		aPtr[i] = new double[size];
	}

	for(int i = 0;i<size;i++)
	{
		for(int j = 0;j<=size;j++)
		{
				aPtr[i][j] = temp[i][j];
		}
	}


	double* bPtr = b.getPtr();
	double* xPtr = x.getPtr();
	//LU decompose

	for(int x = 0;x<size;x++)
	{
		for(int y = 0;y<=x;y++)
		{
			double sum = 0;
			for(int i = 0;i<y;i++)
			{
				sum+=aPtr[i][y]*aPtr[x][i];
			}
			aPtr[x][y] -= sum;
		}

		for(int y = x+1;y<size;y++)
		{
			double sum = 0;
			for(int i = 0;i<x;i++)
			{
				sum+=aPtr[i][y]*aPtr[x][i];
			}
			aPtr[x][y] -=sum;
			if(aPtr[x][x] == 0)
			{
				return 1;
			}
			aPtr[x][y] = aPtr[x][y]/aPtr[x][x];

		}
	}





	//Solve eq
	//L*z = b;

	double* yVect = new double[size];

	for(int y = 0;y<size;y++)
	{
		double sum = 0;
		for(int x = 0;x<y;x++)
		{
			sum += aPtr[x][y]*yVect[x];
		}
		yVect[y] = bPtr[y] - sum;
	}


	for(int y = size-1;y>=0;y--)
	{
		double sum = 0;
		for(int x = y+1;x<size;x++)
		{
			sum+= aPtr[x][y]*xPtr[x];
		}
		xPtr[y] = (yVect[y] - sum)/aPtr[y][y];
	}

	for(int i = 0;i<size;i++)
	{
		delete[] aPtr[i];
	}
	delete[] aPtr;
	delete[] yVect;
	return 0;
}



int Solver::LUSolveCUDA(Matrix& A, Vector& x, Vector& b)
{
	int size = A.getSize();
	double** temp = A.getPtr();
	double** aPtr = new double*[size];
	int error = 0;

	for(int i = 0;i<size;i++)
	{
		aPtr[i] = new double[size];
	}

	double* a2Ptr = new double[size*size];
	double* bPtr = b.getPtr();
	double* xPtr = x.getPtr();

	for(int x = 0;x<size;x++)
	{
		for(int y = 0;y<size;y++)
		{
			a2Ptr[x*size+y] = temp[x][y];
		}
	}

	int* cudaErrorPtr;
	double* cudaAPtr;
	if(hipSuccess != hipMalloc((void**)&cudaAPtr,sizeof(double)*size*size))printf("malloc error");
	if(hipSuccess != hipMalloc((void**)&cudaErrorPtr,sizeof(int)))printf("malloc error");
	if(hipSuccess != hipMemcpy(cudaAPtr,a2Ptr,sizeof(double)*size*size,hipMemcpyHostToDevice))printf("memcpy error");
	if(hipSuccess != hipMemcpy(cudaErrorPtr,&error,sizeof(int),hipMemcpyHostToDevice))printf("memcpy error");




	//LU decompose
	for(int i = 0;i<size;i++)
	{
		int block_size = 512;
		int block_num = ((size-i) + block_size - 1)/block_size;
		computeRow <<<block_num,block_size>>>(cudaAPtr, i, size, cudaErrorPtr);
		computeCol <<<block_num,block_size>>>(cudaAPtr, i, size, cudaErrorPtr);
		if(hipSuccess != hipMemcpy(&error,cudaErrorPtr,sizeof(int),hipMemcpyDeviceToHost))printf("memcpy error");
		if(error != 0) return 1;
	}


	if(hipSuccess != hipMemcpy(a2Ptr,cudaAPtr,sizeof(double)*size*size,hipMemcpyDeviceToHost))printf("memcpy error");

	hipFree(cudaAPtr);
	hipFree(cudaErrorPtr);

	for(int x = 0;x<size;x++)
	{
		for(int y = 0;y<size;y++)
		{
			aPtr[x][y] = a2Ptr[x*size+y];

		}

	}


	//Solve eq
	//L*z = b;

	double* yVect = new double[size];

	for(int y = 0;y<size;y++)
	{
		double sum = 0;
		for(int x = 0;x<y;x++)
		{
			sum += aPtr[x][y]*yVect[x];
		}
		yVect[y] = bPtr[y] - sum;
	}



	for(int y = size-1;y>=0;y--)
	{
		double sum = 0;
		for(int x = y+1;x<size;x++)
		{
			sum+= aPtr[x][y]*xPtr[x];
		}
		xPtr[y] = (yVect[y] - sum)/aPtr[y][y];
	}
	for(int i = 0;i<size;i++)
	{
		delete[] aPtr[i];
	}
	delete[] aPtr;
	delete[] yVect;
	return 0;
}
